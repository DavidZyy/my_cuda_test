#include "hip/hip_runtime.h"
// see [asserts/Screenshot from 2025-01-10 20-33-15.png]()
#include "common.hpp"
#include <cassert>

#define TM  4 
#define TN  4
#define TK  4


/**
 * when TK = 1, a is a col vector, b is a row vector, it can be written as following:
 * @tparam dtype 
 */
template <typename dtype>
__global__ void simt_v1_kernel(const dtype* lhs, const dtype* rhs, dtype* result, size_t M, size_t N, size_t K) {
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;  // Row index
    size_t col = blockIdx.y * blockDim.y + threadIdx.y;  // Column index

    dtype a[TM], b[TN];
    dtype c[TM][TN] = {0};

    for (int k = 0; k < K; k++) {

        // fetch data from global memory
        for (int i = 0; i < TM; i++) {
            a[i] = lhs[(row * TM + i) * K + k];
        }
        for (int j = 0; j < TN; j++) {
            b[j] = rhs[k * N + (col * TN + j)];
        }

        // compute
        for (int i = 0; i < TM; i++) {
            for (int j = 0; j < TN; j++) {
                c[i][j] += a[i] * b[j];
            }
        }
    }

    // write back to global memory
    for (int i = 0; i < TM; i++) {
        for (int j = 0; j < TN; j++) {
            result[(row * TM + i) * N + col * TN + j] = c[i][j];
        }
    }
}

/**
 * more general version
 */
template <typename dtype>
__global__ void simt_v1_kernel2(const dtype* lhs, const dtype* rhs, dtype* result, size_t M, size_t N, size_t K) {
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;  // Row index
    size_t col = blockIdx.y * blockDim.y + threadIdx.y;  // Column index

    dtype a[TM][TK], b[TK][TN];
    dtype c[TM][TN] = {0};

    for (int k = 0; k < K; k += TK) {

        // fetch data from global memory
        for (int i = 0; i < TM; i++) {
            for (int t = 0; t < TK; t++) {
                a[i][t] = lhs[(row * TM + i) * K + k + t];
            }
        }

        for (int j = 0; j < TN; j++) {
            for (int t = 0; t < TK; t++) {
                b[t][j] = rhs[(k + t) * N + (col * TN + j)];
            }
        }

        // compute
        for (int i = 0; i < TM; i++) {
            for (int j = 0; j < TN; j++) {
                // c[i][j] += a[i] * b[j];
                for (int t = 0; t < TK; t++) {
                    c[i][j] += a[i][t] * b[t][j];
                }
            }
        }
    }

    // write back to global memory
    for (int i = 0; i < TM; i++) {
        for (int j = 0; j < TN; j++) {
            result[(row * TM + i) * N + col * TN + j] = c[i][j];
        }
    }
}

/**
 * block matrix multiplication
 * @tparam dtype 
 */
template<typename dtype>
void simt_v1(const dtype* lhs, const dtype* rhs, dtype* result, size_t M, size_t N, size_t K) {
    assert (M % (TM * 16) == 0);
    assert (N % (TN * 16) == 0);
    assert (K % TK == 0);
    dim3 threadsPerBlock(16, 16);  // Define block size (16x16 is a typical choice, can be adjusted)
    dim3 numBlocks((M + (threadsPerBlock.x * TM) - 1) / (threadsPerBlock.x * TM),
                   (N + (threadsPerBlock.y * TN) - 1) / (threadsPerBlock.y * TN));  // Number of blocks

    simt_v1_kernel2<dtype><<<numBlocks, threadsPerBlock>>>(lhs, rhs, result, M, N, K);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}


template void simt_v1<float>(const float* lhs, const float* rhs, float* result, size_t M, size_t N, size_t K);
template void simt_v1<half>(const half* lhs, const half* rhs, half* result, size_t M, size_t N, size_t K);
