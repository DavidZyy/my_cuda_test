#include "hip/hip_runtime.h"
// reference: https://github.com/nicolaswilde/cuda-sgemm
#include "common.hpp"

#include <cassert>
#include <cstddef>
#include <ctime>
#include "omp.h"

template<typename dtype> void simt_v0(const dtype* lhs, const dtype* rhs, dtype* result, size_t M, size_t N, size_t K);
template<typename dtype> void simt_v1(const dtype* lhs, const dtype* rhs, dtype* result, size_t M, size_t N, size_t K);

/***********************************************************************************************************************************************************/
template <typename dtype>
void cpu_sgemm(const dtype* A, const dtype* B, dtype* C, size_t M, size_t N, size_t K) {
    #pragma omp parallel for collapse(2)
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            dtype sum = 0;
            for (int k = 0; k < K; k++) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

/***********************************************************************************************************************************************************/
template <typename dtype>
void cublas_sgemm(const dtype* lhs, const dtype* rhs, dtype* result, size_t M, size_t N, size_t K) {
        hipblasHandle_t handle;
        CUBLAS_CHECK(hipblasCreate(&handle));
        float alpha = 1.0f;
        float beta = 0.0f;
        CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, rhs, N, lhs, K, &beta, result, N));
        CUBLAS_CHECK(hipblasDestroy(handle));
}

/***********************************************************************************************************************************************************/

/**
 * test the max error with cpu and cuda
 */
template <typename dtype, void (*func)(const dtype*, const dtype*, dtype*, size_t, size_t, size_t)>
float testMaxError(int M, int N, int K) {
    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    CUDA_CHECK(hipMalloc(&d_a, size_a));
    CUDA_CHECK(hipMalloc(&d_b, size_b));
    CUDA_CHECK(hipMalloc(&d_c, size_c));
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = rand() / float(RAND_MAX);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

    cpu_sgemm<float>(h_a, h_b, h_c, M, N, K);
    func(d_a, d_b, d_c, M, N, K);

    CUDA_CHECK(hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost));

    float max_error = 0;
    for (int i = 0; i < M * N; i++) {
        float this_error = std::abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = std::max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    free(h_d_c);

    return max_error;
}

template <typename dtype, void (*func)(const dtype*, const dtype*, dtype*, size_t, size_t, size_t)>
float testPerformance(int repeat, size_t M, size_t N, size_t K) {
    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, size_a));
    CUDA_CHECK(hipMalloc(&d_b, size_b));
    CUDA_CHECK(hipMalloc(&d_c, size_c));

    hipEvent_t start, end;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&end));
    CUDA_CHECK(hipEventRecord(start));
    for (int i = 0; i < repeat; i++)
        func(d_a, d_b, d_c, M, N, K);
    CUDA_CHECK(hipEventRecord(end));
    CUDA_CHECK(hipEventSynchronize(end));

    float msec, sec;
    CUDA_CHECK(hipEventElapsedTime(&msec, start, end));
    sec = msec / 1000.0 / repeat;

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));

    return sec;
}

typedef float (*TestMaxErrorFunc)(int, int, int);
TestMaxErrorFunc testMaxErrorFuncs[] = {
    // testMaxError<float, cublas_sgemm<float>>,
    // testMaxError<float, simt_v0<float>>,
    testMaxError<float, simt_v1<float>>
};

void testAllMaxError() {
    int M = 512, N = 512, K = 512;
    for (int j = 0; j < sizeof(testMaxErrorFuncs) / sizeof(TestMaxErrorFunc); j++) {
        float max_error = testMaxErrorFuncs[j](M, N, K);
        printf("M N K = %6d %6d %6d, Max Error = %10.8lf\n", M, N, K, max_error);
    }
}

// Define a type for the function pointers
typedef float (*TestFunc)(int, size_t, size_t, size_t);

// Array of function pointers
TestFunc testFuncs[] = {
    // testPerformance<float, cublas_sgemm<float>>,
    // testPerformance<float, simt_v0<float>>,
    testPerformance<float, simt_v1<float>>
};


void testAllPerformance() {
    const int TESTNUM = 15;
    const int M_list[TESTNUM] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[TESTNUM] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    // const int K_list[TESTNUM] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[TESTNUM] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};

    for (int j = 0; j < sizeof(testFuncs) / sizeof(TestFunc); j++) {
        printf("Test %d\n", j);
        for (int i = 0; i < TESTNUM; i++) {
            int M = M_list[i];
            int N = N_list[i];
            int K = K_list[i];
            float sec = testFuncs[j](10, M, N, K);
            double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / sec; 
            printf("M N K = %6d %6d %6d, AVG Performance = %10.4lf Gflops\n", M, N, K, avg_Gflops);
        }
    }
}

int main() {
    testAllMaxError();
    testAllPerformance();
    return 0;
}
